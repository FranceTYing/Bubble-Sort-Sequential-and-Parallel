#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <fstream>
#include <sys/time.h>

__global__ void even(int *darr, int n)
{
	int k = threadIdx.x + blockIdx.x*blockDim.x;
	int t;
	k= k * 2;
	if (k<=n-2)
	{
		if (darr[k] >darr[k+1])
		{
			t=darr[k];
			darr[k]=darr[k+1];
			darr[k+1]=t;
		}
	
	}
	
}

__global__ void odd(int *darr, int n)
{
	int k = threadIdx.x + blockIdx.x*blockDim.x ;
	int t;
	k= k*2+1;
	if (k<=n-2)
	{
		if (darr[k] >darr[k+1])
		{
			t=darr[k];
			darr[k]=darr[k+1];
			darr[k+1]=t;
		}
	
	}
	
}

void print(int arr[], int row, double t)
{
	int j;
	char filename[32];
	//append file name
	snprintf(filename, sizeof(char) * 32, "OUTPUT_%d.dat", row);
	
	FILE * f = fopen(filename, "w");
	for(j = 0; j < row; j++)
	{
		fprintf(f, "%d\n", arr[j]);
	}
	
	fprintf(f, "\nTime taken = %f seconds. \n", t);
	fclose(f);
}

int main()
{
	int row = 0;
	int temp = 0;
	FILE *file = fopen("data500k.txt", "r");
	int *input = NULL;
	char random;
	double s, e, timeTaken;
	struct timeval t1, t2;
	
	
	if(file == NULL)
	{
		printf("Unable to open file! ");
		return 1;
	}
	else
	{
		printf("Opening file......");
		while(fscanf(file, "%d", &temp) != EOF)
		{
			if(input == NULL)
			{
				input = (int*)malloc(sizeof(temp));
				*input = temp;
			}
			else
			{
				input = (int*)realloc(input, (row+1)*sizeof(temp));
				input[row] = temp;
			}
			row++;
		}
		
		printf("Done read file. ");
		
	}
	fclose(file);
	printf ("\n");
	printf("Press any key to continue......\n");
	random = getchar();
	
	int *copy = (int*)malloc(row * sizeof(int));
	int Line=50000;
	int copyRow;
	while (Line <=row){
	for(copyRow = 0; copyRow < Line; copyRow++)
	{
		copy[copyRow] = input[copyRow];
	}

	int *arr, *darr;
	arr = (int*)malloc (row* sizeof(int));
	int i,copyrow;
	for (copyrow=0; copyrow<copyRow; copyrow++)
	{
		arr[copyrow] =  copy[copyrow];
	}
	
	hipMalloc(&darr, copyrow*sizeof(int));
	hipMemcpy(darr, arr, copyrow*sizeof(int), hipMemcpyHostToDevice);
	//start clock
	
	gettimeofday(&t1, NULL);
		s = (double)(t1.tv_sec + (t1.tv_usec / 1.0e9));

	
	for (i=0; i<=copyrow/2; i++)
	{
		even<< <copyrow/1000, 1000>> >(darr, copyrow);
		odd<< <copyrow/1000, 1000>> >(darr, copyrow);
	}
	gettimeofday(&t2, NULL);
		e = (double)(t2.tv_sec + (t2.tv_usec / 1.0e9));

	timeTaken = e - s;
	hipMemcpy(arr, darr, copyrow*sizeof(int), hipMemcpyDeviceToHost);
	
	printf("\nWhen number of line = %d, ", copyrow);
	printf("Time taken to sort the elements = %f seconds. \n", timeTaken);
	print(arr, copyrow, timeTaken);
	Line= Line + 50000;
	}
	
	getchar();
	return 0;
}